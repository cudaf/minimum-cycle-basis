#include "common.cuh"
#include "utils.h"

hipDeviceProp_t prop;
int device_id;
dim3 dimGrid;
dim3 dimBlock;

extern "C"
void init_cuda()
{
	CudaError(hipGetDevice(&device_id));
	CudaError(hipGetDeviceProperties(&prop,device_id));
}

extern "C"
size_t configure_grid(int start, int end)
{
	dimGrid.x = prop.multiProcessorCount;
	dimGrid.y = 1;
	dimGrid.z = 1;

	dimBlock.x = prop.maxThreadsPerBlock;
	dimBlock.y = 1;
	dimBlock.z = 1;

	size_t sources_to_store;
	if((end-start) < dimGrid.x)
		sources_to_store = end-start;
    else
		sources_to_store = dimGrid.x;

	return sources_to_store;
}
