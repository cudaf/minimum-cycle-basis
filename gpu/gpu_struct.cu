#include "common.cuh"

void gpu_struct::init_memory_setup() {
  CudaError(hipMalloc(&d_non_tree_edges, to_byte_32bit(num_edges)));
  CudaError(
      hipMalloc(&d_edge_offsets,
          to_byte_32bit(chunk_size * original_nodes * nstreams)));
  CudaError(
      hipMalloc(&d_row_offset,
          to_byte_32bit(
              chunk_size * (original_nodes + 1) * nstreams)));
  CudaError(hipMalloc(&d_columns,
  to_byte_32bit(chunk_size * original_nodes) * nstreams));
  CudaError(hipMalloc(&d_precompute_array,
  to_byte_32bit(chunk_size * original_nodes) * nstreams));
  CudaError(hipMalloc(&d_si_vector, to_byte_64bit(size_vector)));
}

void gpu_struct::clear_memory() {
  CudaError(hipFree(d_non_tree_edges));
  CudaError(hipFree(d_edge_offsets));
  CudaError(hipFree(d_row_offset));
  CudaError(hipFree(d_columns));
  CudaError(hipFree(d_precompute_array));
  CudaError(hipFree(d_si_vector));

  destroy_streams();
}

void gpu_struct::init_streams() {
  streams = (hipStream_t*) malloc(nstreams * sizeof(hipStream_t));
  for (int i = 0; i < nstreams; i++)
    CudaError(hipStreamCreate(&(streams[i])));
}

void gpu_struct::destroy_streams() {
  for (int i = 0; i < nstreams; i++)
    CudaError(hipStreamDestroy(streams[i]));

  free(streams);
}

void gpu_struct::init_pitch() {
}

void gpu_struct::calculate_memory() {
  int total_memory_bytes = 0;
  int static_memory_bytes = 0;
  int variable_memory_bytes = 0;

  float size_in_mb = 1024 * 1024;

  static_memory_bytes += to_byte_32bit(num_edges);
  static_memory_bytes += to_byte_64bit(size_vector);

  variable_memory_bytes += to_byte_32bit(
      chunk_size * original_nodes * nstreams);
  variable_memory_bytes += to_byte_32bit(
      chunk_size * (original_nodes + 1) * nstreams);
  variable_memory_bytes += to_byte_32bit(
      chunk_size * original_nodes * nstreams);
  variable_memory_bytes += to_byte_32bit(
      chunk_size * original_nodes * nstreams);

  total_memory_bytes += static_memory_bytes + variable_memory_bytes;

  info->setTotalMemoryUsage((double)total_memory_bytes / size_in_mb);
  info->setStaticMemoryUsage((double)static_memory_bytes /size_in_mb);
  info->setVariableMemoryUsage((double)variable_memory_bytes / size_in_mb);

  printf("Static Memory = %lf mb\n", static_memory_bytes / size_in_mb);
  printf("Variable Memory = %lf mb\n", variable_memory_bytes / size_in_mb);
  printf("total_memory_bytes = %lf mb\n", total_memory_bytes / size_in_mb);

}

void gpu_struct::initialize_memory(gpu_task *host_memory) {

  CudaError(
      hipMemcpy(d_non_tree_edges, host_memory->non_tree_edges_array,
          to_byte_32bit(num_edges), hipMemcpyHostToDevice));

  for (int i = 0; i < nstreams; i++) {
    CudaError(
        hipMemcpy(d_edge_offsets + chunk_size * original_nodes * i,
            host_memory->host_tree->edge_offset[i],
            to_byte_32bit(chunk_size * original_nodes),
            hipMemcpyHostToDevice));

    CudaError(
        hipMemcpy(d_row_offset + chunk_size * (original_nodes + 1) * i,
            host_memory->host_tree->tree_rows[i],
            to_byte_32bit(chunk_size * (original_nodes + 1)),
            hipMemcpyHostToDevice));

    CudaError(
        hipMemcpy(d_columns + chunk_size * original_nodes * i,
            host_memory->host_tree->tree_cols[i],
            to_byte_32bit(chunk_size * original_nodes),
            hipMemcpyHostToDevice));
  }
}

float gpu_struct::copy_support_vector(bit_vector *vector) {
  timer.Start();

  CudaError(
      hipMemcpy(d_si_vector, vector->elements,
          to_byte_64bit(size_vector), hipMemcpyHostToDevice));

  timer.Stop();

  return timer.Elapsed();
}

float gpu_struct::fetch(gpu_task *host_memory) {
  timer.Start();

  for (int i = 0; i < nstreams; i++) {
    CudaError(
        hipMemcpy(host_memory->host_tree->precompute_value[i],
            d_precompute_array + chunk_size * original_nodes * i,
            to_byte_32bit(chunk_size * original_nodes),
            hipMemcpyDeviceToHost));
  }

  timer.Stop();

  return timer.Elapsed();
}

void gpu_struct::transfer_from_asynchronous(int stream_index,
    gpu_task *host_memory,int num_chunk) {

  CudaError(
      hipMemcpyAsync(
          d_edge_offsets + stream_index * chunk_size * original_nodes,
          host_memory->host_tree->edge_offset[num_chunk],
          to_byte_32bit(chunk_size * original_nodes),
          hipMemcpyHostToDevice, streams[stream_index]));

  CudaError(
      hipMemcpyAsync(
          d_row_offset
              + stream_index * chunk_size * (original_nodes + 1),
          host_memory->host_tree->tree_rows[num_chunk],
          to_byte_32bit(chunk_size * (original_nodes + 1)),
          hipMemcpyHostToDevice, streams[stream_index]));

  CudaError(
      hipMemcpyAsync(
          d_columns + stream_index * chunk_size * original_nodes,
          host_memory->host_tree->tree_cols[num_chunk],
          to_byte_32bit(chunk_size * original_nodes),
          hipMemcpyHostToDevice, streams[stream_index]));
}

void gpu_struct::transfer_to_asynchronous(int stream_index,
    gpu_task *host_memory,int num_chunk) {
  CudaError(
      hipMemcpyAsync(
          host_memory->host_tree->precompute_value[num_chunk],
          d_precompute_array
              + stream_index * chunk_size * original_nodes,
          to_byte_32bit(chunk_size * original_nodes),
          hipMemcpyDeviceToHost, streams[stream_index]));
}

float gpu_struct::process_shortest_path(gpu_task *host_memory,
    bool multiple_transfer) {
  timer.Start();

  for (int i = 0; i < num_chunks; i++) {

    int start = (i%nstreams) * chunk_size;
    int end = (i%nstreams + 1) * chunk_size;

    if (multiple_transfer)
      transfer_from_asynchronous(i%nstreams, host_memory, i);

    Kernel_init_edges_helper(start, end, i%nstreams);

    Kernel_multi_search_helper(start, end, i%nstreams);

    transfer_to_asynchronous(i%nstreams, host_memory, i);
  }

  CudaError(hipDeviceSynchronize());

  timer.Stop();

  return timer.Elapsed();
}
