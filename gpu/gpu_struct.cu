#include "hip/hip_runtime.h"
#include "common.cuh"

void gpu_struct::init_memory_setup() {
	CudaError(hipMalloc(&d_non_tree_edges, to_byte_32bit(num_edges)));
	CudaError(
			hipMalloc(&d_edge_offsets,
					to_byte_32bit(chunk_size * original_nodes)));
	CudaError(
			hipMalloc(&d_row_offset,
					to_byte_32bit(chunk_size * (original_nodes + 1))));
	CudaError(
			hipMalloc(&d_columns, to_byte_32bit(chunk_size * original_nodes)));
	CudaError(
			hipMalloc(&d_precompute_array,
					to_byte_32bit(chunk_size * original_nodes)));
	CudaError(hipMalloc(&d_si_vector, to_byte_64bit(size_vector)));
}

void gpu_struct::init_streams() {
	streams = (hipStream_t*) malloc(nstreams * sizeof(hipStream_t));
	for (int i = 0; i < nstreams; i++)
		CudaError(hipStreamCreate(&(streams[i])));
}

void gpu_struct::destroy_streams() {
	for (int i = 0; i < nstreams; i++)
		CudaError(hipStreamDestroy(streams[i]));

	free(streams);
}

void gpu_struct::clear_memory() {
	CudaError(hipFree(d_non_tree_edges));
	CudaError(hipFree(d_edge_offsets));
	CudaError(hipFree(d_row_offset));
	CudaError(hipFree(d_columns));
	CudaError(hipFree(d_precompute_array));
	CudaError(hipFree(d_si_vector));

	destroy_streams();
}

void gpu_struct::init_pitch() {
}

void gpu_struct::calculate_memory() {
	int total_memory_bytes = 0;
	int static_memory_bytes = 0;
	int variable_memory_bytes = 0;

	float size_in_mb = 1024 * 1024;

	static_memory_bytes += to_byte_32bit(num_edges);
	static_memory_bytes += to_byte_64bit(size_vector);

	variable_memory_bytes += to_byte_32bit(chunk_size * original_nodes);
	variable_memory_bytes += to_byte_32bit(chunk_size * (original_nodes + 1));
	variable_memory_bytes += to_byte_32bit(chunk_size * original_nodes);
	variable_memory_bytes += to_byte_32bit(chunk_size * original_nodes);
	variable_memory_bytes += to_byte_32bit(fvs_size);

	total_memory_bytes += static_memory_bytes + variable_memory_bytes;

	printf("Static Memory = %lf mb\n", static_memory_bytes / size_in_mb);
	printf("Variable Memory = %lf mb\n", variable_memory_bytes / size_in_mb);
	printf("total_memory_bytes = %lf mb\n", total_memory_bytes / size_in_mb);

}

void gpu_struct::initialize_memory(gpu_task *host_memory) {
	CudaError(
			hipMemcpy(d_non_tree_edges, host_memory->non_tree_edges_array,
					to_byte_32bit(num_edges), hipMemcpyHostToDevice));

	CudaError(
			hipMemcpy(d_edge_offsets, host_memory->host_tree->edge_offset[0],
					to_byte_32bit(chunk_size * original_nodes),
					hipMemcpyHostToDevice));

	CudaError(
			hipMemcpy(d_row_offset, host_memory->host_tree->tree_rows[0],
					to_byte_32bit(chunk_size * (original_nodes + 1)),
					hipMemcpyHostToDevice));

	CudaError(
			hipMemcpy(d_columns, host_memory->host_tree->tree_cols[0],
					to_byte_32bit(chunk_size * original_nodes),
					hipMemcpyHostToDevice));
}

float gpu_struct::copy_support_vector(bit_vector *vector) {
	timer.Start();

	CudaError(
			hipMemcpy(d_si_vector, vector->elements,
					to_byte_64bit(size_vector), hipMemcpyHostToDevice));

	timer.Stop();

	return timer.Elapsed();
}

float gpu_struct::fetch(gpu_task *host_memory) {
	timer.Start();

	CudaError(
			hipMemcpy(host_memory->host_tree->precompute_value[0],
					d_precompute_array,
					to_byte_32bit(chunk_size * original_nodes),
					hipMemcpyDeviceToHost));

	timer.Stop();

	return timer.Elapsed();
}

void gpu_struct::transfer_from_asynchronous(int start, int end, int fvs_index,
		int stream_index, gpu_task *host_memory) {
	CudaError(
			hipMemcpyAsync(
					d_edge_offsets + stream_index * chunk_size * original_nodes,
					host_memory->host_tree->edge_offset[fvs_index],
					to_byte_32bit(
							min(end - start, chunk_size) * original_nodes),
					hipMemcpyHostToDevice, streams[stream_index]));

	CudaError(
			hipMemcpyAsync(
					d_row_offset
							+ stream_index * chunk_size * (original_nodes + 1),
					host_memory->host_tree->tree_rows[fvs_index],
					to_byte_32bit(
							min(end - start, chunk_size)
									* (original_nodes + 1)),
					hipMemcpyHostToDevice, streams[stream_index]));

	CudaError(
			hipMemcpyAsync(
					d_columns + stream_index * chunk_size * original_nodes,
					host_memory->host_tree->tree_cols[fvs_index],
					to_byte_32bit(
							min(end - start, chunk_size) * original_nodes),
					hipMemcpyHostToDevice, streams[stream_index]));
}

void gpu_struct::transfer_to_asynchronous(int start, int end, int fvs_index,
		int stream_index, gpu_task *host_memory) {
	CudaError(
			hipMemcpyAsync(host_memory->host_tree->precompute_value[fvs_index],
					d_precompute_array
							+ stream_index * chunk_size * original_nodes,
					to_byte_32bit(
							min(end - start, chunk_size) * original_nodes),
					hipMemcpyDeviceToHost, streams[stream_index]));
}
