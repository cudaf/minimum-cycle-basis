#include "common.cuh"

void gpu_struct::init_memory_setup()
{
	CudaError(hipMalloc(&d_non_tree_edges,to_byte_32bit(num_edges)));
	CudaError(hipMalloc(&d_edge_offsets,to_byte_32bit(chunk_size * original_nodes)));
	CudaError(hipMalloc(&d_row_offset,to_byte_32bit(chunk_size * (original_nodes + 1))));
	CudaError(hipMalloc(&d_columns,to_byte_32bit(chunk_size * original_nodes)));
	CudaError(hipMalloc(&d_precompute_array,to_byte_32bit(chunk_size * original_nodes)));
	CudaError(hipMalloc(&d_si_vector,to_byte_64bit(size_vector)));
	CudaError(hipMalloc(&d_fvs_vertices,to_byte_32bit(fvs_size)));
}

void gpu_struct::init_streams()
{
	streams = (hipStream_t*) malloc(nstreams*sizeof(hipStream_t));
	for(int i=0;i<nstreams;i++)
		CudaError(hipStreamCreate(&(streams[i])));
}

void gpu_struct::destroy_streams()
{
	for(int i=0;i<nstreams;i++)
		CudaError(hipStreamDestroy(streams[i]));

	free(streams);
}

gpu_struct::~gpu_struct()
{
	CudaError(hipFree(d_non_tree_edges));
	CudaError(hipFree(d_edge_offsets));
	CudaError(hipFree(d_row_offset));
	CudaError(hipFree(d_columns));
	CudaError(hipFree(d_precompute_array));
	CudaError(hipFree(d_si_vector));
	CudaError(hipFree(d_fvs_vertices));

	destroy_streams();
}

void gpu_struct::init_pitch()
{
}

void gpu_struct::calculate_memory()
{
	int total_memory_bytes = 0;
	int static_memory_bytes = 0;
	int variable_memory_bytes = 0;

	float size_in_mb = 1024*1024;

	static_memory_bytes += to_byte_32bit(num_edges);
	static_memory_bytes += to_byte_64bit(size_vector);

	variable_memory_bytes += to_byte_32bit(chunk_size * original_nodes);
	variable_memory_bytes += to_byte_32bit(chunk_size * (original_nodes + 1));
	variable_memory_bytes += to_byte_32bit(chunk_size * original_nodes);
	variable_memory_bytes += to_byte_32bit(chunk_size * original_nodes);
	variable_memory_bytes += to_byte_32bit(fvs_size);

	total_memory_bytes += static_memory_bytes + variable_memory_bytes;

	printf("Static Memory = %lf mb\n",static_memory_bytes / size_in_mb);
	printf("Variable Memory = %lf mb\n",variable_memory_bytes / size_in_mb);
	printf("total_memory_bytes = %lf mb\n",total_memory_bytes / size_in_mb);

}

void gpu_struct::initialize_memory(gpu_task *host_memory)
{
	CudaError(hipMemcpy(d_non_tree_edges,host_memory->non_tree_edges_array,
			  to_byte_32bit(num_edges),hipMemcpyHostToDevice));

	CudaError(hipMemcpy(d_edge_offsets,host_memory->host_tree->edge_offset[0],
			  to_byte_32bit(chunk_size * original_nodes),hipMemcpyHostToDevice));

	CudaError(hipMemcpy(d_row_offset,host_memory->host_tree->tree_rows[0],
			  to_byte_32bit(chunk_size * (original_nodes + 1)),hipMemcpyHostToDevice));

	CudaError(hipMemcpy(d_columns,host_memory->host_tree->tree_cols[0],
			  to_byte_32bit(chunk_size * original_nodes),hipMemcpyHostToDevice));

	CudaError(hipMemcpy(d_precompute_array,host_memory->host_tree->precompute_value[0],
			  to_byte_32bit(chunk_size * original_nodes),hipMemcpyHostToDevice));

	CudaError(hipMemcpy(d_fvs_vertices,host_memory->fvs_array,
			  to_byte_32bit(fvs_size),hipMemcpyHostToDevice));

}

float gpu_struct::copy_support_vector(bit_vector *vector)
{
	timer.Start();

	CudaError(hipMemcpy(d_si_vector,vector->elements,to_byte_64bit(size_vector),hipMemcpyHostToDevice));

	timer.Stop();

	return timer.Elapsed();
}

float gpu_struct::fetch(gpu_task *host_memory)
{
	timer.Start();

	CudaError(hipMemcpy(host_memory->host_tree->precompute_value[0],d_precompute_array,
			  to_byte_32bit(chunk_size * original_nodes),hipMemcpyDeviceToHost));

	timer.Stop();

	return timer.Elapsed();
}
