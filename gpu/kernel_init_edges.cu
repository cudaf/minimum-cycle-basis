#include "hip/hip_runtime.h"
#include "gpu_struct.cuh"
#include "common.cuh"


template<typename T>
__device__  __forceinline__
 const T* get_pointer_const(const T *data, int node, int nodes, int chunk_size, int stream) {
  return (data + (stream * chunk_size * nodes) + (node * nodes));
}

template<typename T>
__device__  __forceinline__
T* get_pointer(T* data, int node, int nodes, int chunk_size, int stream) {
  return (data + (stream * chunk_size * nodes) + (node * nodes));
}

__device__ __forceinline__
unsigned getBit(unsigned long long val, int pos) {
  unsigned long long ret;
  asm("bfe.u64 %0, %1, %2, 1;" : "=l"(ret) : "l"(val), "r"(pos));
  return (unsigned) ret;
}

__global__
void __kernel_init_edge(const int* __restrict__ d_non_tree_edges,
    const int* __restrict__ d_edge_offsets, int *d_precompute_array,
    const int* __restrict__ d_fvs_vertices,
    const uint64_t *d_si_vector, int start, int end,
    int stream, int chunk_size, int original_nodes, int size_vector,
    int fvs_size, int num_non_tree_edges, int num_edges) {
  int si_index = -1;

  uint64_t si_value;
  int src_index = blockIdx.x + start;

  if (src_index >= end)
    return;

  int *d_row = get_pointer(d_precompute_array, src_index - start,
      original_nodes, chunk_size, stream);
  const int* __restrict__ d_edge = get_pointer_const(d_edge_offsets,
      src_index - start, original_nodes, chunk_size, stream);

  for (int edge_index = threadIdx.x; edge_index < original_nodes;
      edge_index += blockDim.x) {
    int edge_offset = __ldg(&d_edge[edge_index]);
    //tree edges
    if (edge_offset >= 0) {
      int non_tree_edge_loc = __ldg(&d_non_tree_edges[edge_offset]);

      //non_tree_edge
      if (non_tree_edge_loc >= 0) {
        int p_idx = non_tree_edge_loc >> 6;
        if (si_index != p_idx) {
          si_index = p_idx;
          si_value = __ldg(&d_si_vector[si_index]);
        }
        d_row[edge_index] = getBit(si_value, non_tree_edge_loc & 63);
      } else
        //tree edge
        d_row[edge_index] = 0;
    } else {
      d_row[edge_index] = 0;
    }
  }
}

/**
 * @brief
 * This method is used to invoke a kernel whose function is defined in the details section.
 * @details
 * This method invokes a Kernel. The Kernel's task is to parallely do the following things
 * in the order.
 * a) For each source vertex between start and end (15 at a time(grid dimension)). We fill
 *    the precompute_array edges
 * b)The precompute array is filled in the following way.
 *   If the edge is a tree edge in the original spanning tree. then its value is 0.
 *   else if Si contains 1 in the corresponding non-tree edge position then 1 else 0.
 *
 * @param start index of vertex from 0 - fvs_size - 2
 * @param end index of vertex from 1 to fvs_size - 1
 * @param stream 0 or 1
 */
void gpu_struct::Kernel_init_edges_helper(int start, int end, int stream) {
  int total_length = end - start;

  __kernel_init_edge<<<total_length, 512, 0, streams[stream]>>>(
      d_non_tree_edges, d_edge_offsets, d_precompute_array,
      d_fvs_vertices, d_si_vector, start, end, stream, chunk_size,
      original_nodes, size_vector, fvs_size, num_non_tree_edges, num_edges);
}
